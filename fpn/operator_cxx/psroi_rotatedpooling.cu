#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2017 by Contributors
 * Copyright (c) 2017 Microsoft
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file psroi_pooling.cu
 * \brief psroi pooling operator
 * \author Yi Li, Tairui Chen, Guodong Zhang, Haozhi Qi, Jifeng Dai
 * modified by Jian Ding
*/
#include "./psroi_rotatedpooling-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>
#include "../../common/cuda_utils.h"
#include "../mxnet_op.h"

#define PSROIROTATEDPOOLING_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)
#define CUDA_KERNEL_LOOP(i, n) \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
      i < (n); \
      i += blockDim.x * gridDim.x)

namespace mshadow {
namespace cuda {

template <typename DType>
__global__ void PSROIROTATEDPoolForwardKernel(
  const int count,
  const DType* bottom_data,
  const DType spatial_scale,
  const int channels,
  const int height, const int width,
  const int pooled_height, const int pooled_width,
  const DType* bottom_rois,
  const int output_dim,
  const int group_size,
  DType* top_data) {
  CUDA_KERNEL_LOOP(index, count) {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const DType* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];
    DType roi_xc = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale;
    DType roi_yc = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale;
    DType roi_w = static_cast<DType>(round(offset_bottom_rois[3])) * spatial_scale;
    DType roi_h = static_cast<DType>(round(offset_bottom_rois[4])) * spatial_scale;
    DType Theta = static_cast<DType>(offset_bottom_rois[5]);

    DType cosTheta = cos(Theta);
    DType sinTheta = sin(Theta);

    // Force too small ROIs to be 1x1
    DType roi_width = max(roi_w, 1.);  // avoid 0
    DType roi_height = max(roi_h, 1.);

    // Compute w and h at bottom
    DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
    DType bin_size_w = roi_width / static_cast<DType>(pooled_width);

    int hstart = floor(static_cast<DType>(ph) * bin_size_h);
    int wstart = floor(static_cast<DType>(pw)* bin_size_w);
    int hend = ceil(static_cast<DType>(ph + 1) * bin_size_h);
    int wend = ceil(static_cast<DType>(pw + 1) * bin_size_w);
    // Add roi offsets and clip to input boundaries
    // hstart = min(max(hstart, 0), roi_h);
    // hend = min(max(hend, 0), roi_h);
    // wstart = min(max(wstart, 0), roi_w);
    // wend = min(max(wend, 0), roi_w);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    int gw = floor(static_cast<DType>(pw)* group_size / pooled_width);
    int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
    gw = min(max(gw, 0), group_size - 1);
    gh = min(max(gh, 0), group_size - 1);
    int c = (ctop*group_size + gh)*group_size + gw;

    const DType* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;
    DType out_sum = 0;
    float half_w = (float)(roi_w)/2.0;
    float half_h = (float)(roi_h)/2.0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        float xx = cosTheta*((float)(w) - half_w) - sinTheta*((float)(h) - half_h) + roi_xc;
        float yy = sinTheta*((float)(w) - half_w) + cosTheta*((float)(h) - half_h) + roi_yc; 
        int xint = (int)(round(xx));
        int yint = (int)(round(yy));
        
        if (xint >= width || xint < 0 || yint>=height || yint < 0)
          continue;

        int bottom_index = yint*width + xint;
        out_sum += offset_bottom_data[bottom_index];
      }
    }

    DType bin_area = (hend - hstart)*(wend - wstart);
    top_data[index] = is_empty? (DType)0. : out_sum/bin_area;
  }
}

template<typename DType>
inline void PSROIROTATEDPoolForward(const Tensor<gpu, 4, DType> &out,
                           const Tensor<gpu, 4, DType> &data,
                           const Tensor<gpu, 2, DType> &bbox,
                           const float spatial_scale,
                           const int output_dim_,
                           const int group_size_) {
  const DType *bottom_data = data.dptr_;
  const DType *bottom_rois = bbox.dptr_;
  DType *top_data = out.dptr_;
  const int count = out.shape_.Size();
  const int channels = data.size(1);
  const int height = data.size(2);
  const int width = data.size(3);
  const int pooled_height = out.size(2);
  const int pooled_width = out.size(3);
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  PSROIROTATEDPoolForwardKernel<DType> << <mxnet::op::mxnet_op::cuda_get_num_blocks(count),
    kBaseThreadNum, 0, stream >> >(
      count, bottom_data, spatial_scale, channels, height, width,
      pooled_height, pooled_width, bottom_rois, output_dim_, group_size_, top_data);
  PSROIROTATEDPOOLING_CUDA_CHECK(hipPeekAtLastError());
}


template <typename DType>
__global__ void PSROIROTATEDPoolBackwardAccKernel(
  const int count,
  const DType* top_diff,
  const int num_rois,
  const DType spatial_scale,
  const int channels,
  const int height, const int width,
  const int pooled_height, const int pooled_width,
  const int group_size,
  const int output_dim,
  DType* bottom_diff,
  const DType* bottom_rois) {
  CUDA_KERNEL_LOOP(index, count) {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const DType* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];
    DType roi_xc = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale;
    DType roi_yc = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale;
    DType roi_w = static_cast<DType>(round(offset_bottom_rois[3])) * spatial_scale;
    DType roi_h = static_cast<DType>(round(offset_bottom_rois[4])) * spatial_scale;
    DType Theta = static_cast<DType>(offset_bottom_rois[5]);

    DType cosTheta = cos(Theta);
    DType sinTheta = sin(Theta);

    // Force too small ROIs to be 1x1
    DType roi_width = max(roi_w, 1.0);  // avoid 0
    DType roi_height = max(roi_h, 1.0);

    // Compute w and h at bottom
    DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
    DType bin_size_w = roi_width / static_cast<DType>(pooled_width);

    int hstart = floor(static_cast<DType>(ph)* bin_size_h);
    int wstart = floor(static_cast<DType>(pw)* bin_size_w);
    int hend = ceil(static_cast<DType>(ph + 1) * bin_size_h);
    int wend = ceil(static_cast<DType>(pw + 1) * bin_size_w);
    // Add roi offsets and clip to input boundaries
    // hstart = min(max(hstart, 0), roi_h);
    // hend = min(max(hend, 0), roi_h);
    // wstart = min(max(wstart, 0), roi_w);
    // wend = min(max(wend, 0), roi_w);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Compute c at bottom
    int gw = floor(static_cast<DType>(pw)* group_size / pooled_width);
    int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
    gw = min(max(gw, 0), group_size - 1);
    gh = min(max(gh, 0), group_size - 1);
    int c = (ctop*group_size + gh)*group_size + gw;
    DType* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;
    DType bin_area = (hend - hstart)*(wend - wstart);
    DType diff_val = is_empty ? (DType)0. : top_diff[index] / bin_area;

    float half_w = (float)(roi_w)/2.0;
    float half_h = (float)(roi_h)/2.0;

    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        float xx = cosTheta*((float)(w)-half_w) - sinTheta*((float)(h)-half_h) + roi_xc;
        float yy = sinTheta*((float)(w)-half_w) + cosTheta*((float)(h)-half_h) + roi_yc;
        int xint = (int)(round(xx));
        int yint = (int)(round(yy));

        if (xint>=width || xint<0 || yint>=height || yint<0)
          continue;

        int bottom_index = yint*width + xint;
        atomicAdd(offset_bottom_diff + bottom_index, diff_val);
      }
    }
  }
}


template<typename DType>
inline void PSROIROTATEDPoolBackwardAcc(const Tensor<gpu, 4, DType> &in_grad,
                            const Tensor<gpu, 4, DType> &out_grad,
                            const Tensor<gpu, 2, DType> &bbox,
                            const float spatial_scale,
                            const int output_dim_,
                            const int group_size_) {
  // LOG(INFO) << "PSROIROTATEDPoolBackward";
  const DType *top_diff = out_grad.dptr_;
  const DType *bottom_rois = bbox.dptr_;
  DType *bottom_diff = in_grad.dptr_;
  const int count = out_grad.shape_.Size();
  const int num_rois = bbox.size(0);
  const int channels = in_grad.size(1);
  const int height = in_grad.size(2);
  const int width = in_grad.size(3);
  const int pooled_height = out_grad.size(2);
  const int pooled_width = out_grad.size(3);
  hipStream_t stream = Stream<gpu>::GetStream(in_grad.stream_);
  PSROIROTATEDPoolBackwardAccKernel<DType> << <mxnet::op::mxnet_op::cuda_get_num_blocks(count),
    kBaseThreadNum, 0, stream >> >(
      count, top_diff, num_rois, spatial_scale, channels, height, width,
      pooled_height, pooled_width, group_size_, output_dim_, bottom_diff, bottom_rois);
  PSROIROTATEDPOOLING_CUDA_CHECK(hipPeekAtLastError());
}

}  // namespace cuda

template<typename DType>
inline void PSROIROTATEDPoolForward(const Tensor<gpu, 4, DType> &out,
                           const Tensor<gpu, 4, DType> &data,
                           const Tensor<gpu, 2, DType> &bbox,
                           const float spatial_scale,
                           const int output_dim_,
                           const int group_size_) {
  cuda::PSROIROTATEDPoolForward(out, data, bbox, spatial_scale, output_dim_, group_size_);
}

template<typename DType>
inline void PSROIROTATEDPoolBackwardAcc(const Tensor<gpu, 4, DType> &in_grad,
                            const Tensor<gpu, 4, DType> &out_grad,
                            const Tensor<gpu, 2, DType> &bbox,
                            const float spatial_scale,
                            const int output_dim_,
                            const int group_size_) {
  cuda::PSROIROTATEDPoolBackwardAcc(in_grad, out_grad, bbox, spatial_scale, output_dim_, group_size_);
}

}  // namespace mshadow


namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(PSROIROTATEDPoolingParam param, int dtype) {
  Operator* op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new PSROIROTATEDPoolingOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
